#include "hip/hip_runtime.h"
#include <iostream>

#include "render_option.h"
#include "ray.cuh"
#include "random.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

__global__ void dummyKernel(hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(1984, i, 0, &rand_state[i]);

    printf("Random number GPU: %f\n", randf(&rand_state[i]));

    printf("Random vec3 GPU: %f %f %f\n", randVec3(&rand_state[i]).x(), randVec3(&rand_state[i]).y(), randVec3(&rand_state[i]).z());

}

int main(int argc, char** argv) {

    std::cout << "Random number CPU: " << randf() << "\n";

    std::cout << "Random vec3 CPU: " << randVec3() << "\n";

    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, sizeof(hiprandState)));

    dummyKernel<<<1, 1>>>(d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());


    return 0;
}
